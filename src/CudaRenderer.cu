#include "hip/hip_runtime.h"
#include "CudaRenderer.hpp"

#include <GL/glew.h>
#include <GL/gl.h>

#include <glm/gtx/component_wise.hpp>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Utils.hpp"
#include "Triangle.hpp"


#define BLOCKWIDTH 8
#define INTERSECT_EPSILON 0.0000001f
#define OFFSET_EPSILON 0.00001f
#define BIGT 99999.f
#define SHADOWSAMPLING 8
#define SECONDARY_RAYS 3
#define AIR_INDEX 1.f

#define REFLECTIVE_BIT 0x80000000
#define REFRACTIVE_BIT 0x40000000
#define INSIDE_BIT 0x20000000

#define LEFT_HIT_BIT 0x80000000
#define RIGHT_HIT_BIT 0x40000000

#define PATH_TRACE_BOUNCES 6

inline __device__ float fresnelReflectioncoefficient(const float sin2t, const float cosi, const float idx1, const float idx2)
{
  const float cost = sqrt(1 - sin2t);

  float Rs = (idx1 * cosi - idx2 * cost) / (idx1 * cosi + idx2 * cost);
  Rs = Rs * Rs;

  float Rp = (idx2 * cosi - idx1 * cost) / (idx2 * cosi + idx1 * cost);
  Rp = Rp * Rp;

  return (Rs + Rp) * 0.5f;
}

__device__ glm::fmat3 getBasis(const glm::fvec3 n) {

  glm::fmat3 R;

  glm::fvec3 Q = n;
  const glm::fvec3 absq = glm::abs(Q);
  float absqmin = glm::compMin(absq);

  for (int i = 0; i < 3; ++i) {
    if (absq[i] == absqmin) {
      Q[i] = 1;
      break;
    }
  }

  glm::fvec3 T = glm::normalize(glm::cross(Q, n));
  glm::fvec3 B = glm::normalize(glm::cross(n, T));

  R[0] = T;
  R[1] = B;
  R[2] = n;

  return R;
}

__device__ bool bboxIntersect(const AABB box, const glm::fvec3 origin, const glm::fvec3 inverseDirection, float& t)
{
  glm::fvec3 tmin(-BIGT), tmax(BIGT);

  const glm::fvec3 tdmin = (box.min - origin) * inverseDirection;
  const glm::fvec3 tdmax = (box.max - origin) * inverseDirection;

  tmin = glm::min(tdmin, tdmax);
  tmax = glm::max(tdmin, tdmax);

  const float tmind = glm::compMax(tmin);
  const float tmaxd = glm::compMin(tmax);

  t = fminf(tmind, tmaxd);

  return tmaxd >= tmind && !(tmaxd < 0.f && tmind < 0.f);
}

__device__ void debug_fvec3(const glm::fvec3 v)
{
  printf("%f %f %f\n", v.x, v.y, v.z);
}

inline __device__ glm::fvec3 reflectionDirection(const glm::vec3 normal, const glm::vec3 incoming) {

  const float cosT = glm::dot(incoming, normal);

  return incoming - 2 * cosT * normal;
}

inline __device__ glm::fvec3 refractionDirection(const float cosInAng, const float sin2t, const glm::vec3 normal, const glm::vec3 incoming, const float index1, const float index2)
{
    return index1 / index2 * incoming + (index1 / index2 * cosInAng - sqrt(1 - sin2t)) * normal;
}

__device__ bool rayTriangleIntersection(const Ray ray, const Triangle& triangle, float& t, glm::fvec2& uv)
{
  /* Möller-Trumbore algorithm
   * https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
   */

  // TODO: Experiment with __ldg

  const glm::vec3 vertex0 = triangle.vertices[0].p;

  const glm::fvec3 edge1 = triangle.vertices[1].p - vertex0;
  const glm::fvec3 edge2 = triangle.vertices[2].p - vertex0;

  const glm::fvec3 h = glm::cross(ray.direction, edge2);
  const float a = glm::dot(edge1, h);

  if (a > -INTERSECT_EPSILON && a < INTERSECT_EPSILON)
    return false;

  const float f = __fdividef(1.f, a);
  const glm::fvec3 s = ray.origin - vertex0;
  const float u = f * glm::dot(s, h);

  if (u < 0.f || u > 1.0f)
    return false;

  const glm::fvec3 q = glm::cross(s, edge1);
  const float v = f * glm::dot(ray.direction, q);

  if (v < 0.0 || u + v > 1.0)
    return false;

  t = f * glm::dot(edge2, q);

  if (t > INTERSECT_EPSILON)
  {
    uv = glm::fvec2(u, v);
    return true;
  }
  else
    return false;
}

enum HitType
{
    ANY,
    CLOSEST
};

template <bool debug, const HitType hitType>
__device__
RaycastResult rayCast(const Ray ray, const Node* bvh, const Triangle* triangles, const float maxT)
{
  float tMin = maxT;
  int minTriIdx = -1;
  glm::fvec2 minUV;
  RaycastResult result;
  const glm::fvec3 inverseDirection = glm::fvec3(1.f) / ray.direction;

  int ptr = 0;
  unsigned int stack[8] { 0 };
  int i = -1;
  float t = 0;
  glm::fvec2 uv;
  bool getNextNode = true;

  while (ptr >= 0)
  {
    unsigned int currentNodeIdx = stack[ptr];
    Node currentNode = bvh[currentNodeIdx];


    if (currentNode.rightIndex == -1)
    {
      getNextNode = false;

      if (debug)
      {
        const AABB b = currentNode.bbox;
        printf("\nHit bbox %d:\n", currentNodeIdx);
        printf("min: %f %f %f\n", b.min[0], b.min[1], b.min[2]);
        printf("max: %f %f %f\n", b.max[0], b.max[1], b.max[2]);
        printf("StartIdx: %d, endIdx: %d, nTris: %d\n\n", currentNode.startTri, currentNode.startTri + currentNode.nTri, currentNode.nTri);
      }

      if (i >= currentNode.startTri && i < currentNode.startTri + currentNode.nTri)
      {
        if (rayTriangleIntersection(ray, triangles[i], t, uv))
        {
          if (debug)
            printf("Hit triangle %d\n", i);

          if(t < tMin)
          {
            tMin = t;
            minTriIdx = i;
            minUV = uv;

            if (hitType == HitType::ANY)
              break;
          }
        }

        ++i;

        if (i >= currentNode.startTri + currentNode.nTri)
          getNextNode = true;

      }else
      {
        i = currentNode.startTri;
      }

    }else
    {
      const AABB leftBox = bvh[stack[ptr] + 1].bbox;
      const AABB rightBox = bvh[currentNode.rightIndex].bbox;

      float leftt, rightt;

      unsigned int hitMask = bboxIntersect(leftBox, ray.origin, inverseDirection, leftt) ? LEFT_HIT_BIT : 0x00;
      hitMask = bboxIntersect(rightBox, ray.origin, inverseDirection, rightt) ? hitMask | RIGHT_HIT_BIT : hitMask;

      // TODO: Push closer one last, don't intersect if closest hit is closer than box
      if ((hitMask & LEFT_HIT_BIT) != 0x00 && leftt < tMin)
      {
        stack[ptr] = currentNodeIdx + 1;
        ++ptr;
      }

      if ((hitMask & RIGHT_HIT_BIT) != 0x00 && rightt < tMin)
      {
        stack[ptr] = currentNode.rightIndex;
        ++ptr;
      }
    }

    if (getNextNode)
    {
      --ptr;
      i = -1;
    }

  }

  if (minTriIdx == -1)
    return result;

  result.point = ray.origin + ray.direction * tMin;
  result.t = tMin;
  result.triangleIdx = minTriIdx;
  result.uv = minUV;

  if (debug)
    printf("///////////////////\n\n");

  return result;
}


template<unsigned int samples, typename hiprandState>
__device__ glm::fvec3 areaLightShading(const glm::fvec3 interpolatedNormal, const Light& light, const Node* bvh, const RaycastResult& result, const Triangle* triangles, hiprandState& curandState1, hiprandState& curandState2)
{
  glm::fvec3 brightness(0.f);

  //if (!light.isEnabled()) // Surprisingly slow
  //  return brightness;

  const glm::fvec3 shadowRayOrigin = result.point + interpolatedNormal * OFFSET_EPSILON;

  glm::fvec3 lightSamplePoint;
  float pdf;

  const glm::fvec3 emission = light.getEmission();

  // TODO: Unroll using templates
  for (unsigned int i = 0; i < samples; ++i)
  {
    light.sample(pdf, lightSamplePoint, curandState1, curandState2);

    const glm::fvec3 shadowRayDir = lightSamplePoint - shadowRayOrigin;

    const float maxT = glm::length(shadowRayDir); // Distance to the light
    const glm::fvec3 shadowRayDirNormalized = shadowRayDir / maxT;

    const Ray shadowRay(shadowRayOrigin, shadowRayDirNormalized);

    const RaycastResult shadowResult = rayCast<false, HitType::ANY>(shadowRay, bvh, triangles, maxT);

    if ((shadowResult && shadowResult.t >= maxT + OFFSET_EPSILON) || !shadowResult)
    {
      const float cosOmega = __saturatef(glm::dot(shadowRayDirNormalized, interpolatedNormal));
      const float cosL = __saturatef(glm::dot(-shadowRayDirNormalized, light.getNormal()));

      brightness += __fdividef(1.f, (maxT * maxT * pdf)) * emission * cosL * cosOmega;
    }
  }

  brightness /= samples;

  return brightness;
}

__device__ inline constexpr unsigned int cpow(const unsigned int base, const unsigned int exponent)
{
    return (exponent == 0) ? 1 : (base * cpow(base, exponent - 1));
}

struct RaycastTask
{
  Ray outRay;
  unsigned short levelsLeft;
  glm::fvec3 filter;
};

template <bool debug, typename curandStateType>
__device__ glm::fvec3 rayTrace(\
    const Node* bvh, \
    const Ray& ray, \
    const Triangle* triangles, \
    const Camera camera, \
    const Material* materials, \
    const unsigned int* triangleMaterialIds, \
    const Light light, \
    curandStateType& curandState1, \
    curandStateType& curandState2, \
    glm::fvec3* hitPoints = nullptr)
{
  constexpr unsigned int stackSize = cpow(2, SECONDARY_RAYS);
  RaycastTask stack[stackSize];
  glm::fvec3 color(0.f);
  int stackPtr = 0;
  int posPtr = 0; // Probably optimized away when not used

  // Primary ray
  stack[stackPtr].outRay = ray;
  stack[stackPtr].levelsLeft = SECONDARY_RAYS;
  stack[stackPtr].filter = glm::fvec3(1.f);
  ++stackPtr;

  while (stackPtr > 0)
  {
    --stackPtr;

    const RaycastTask currentTask = stack[stackPtr];
    const RaycastResult result = rayCast<false, HitType::CLOSEST>(currentTask.outRay, bvh, triangles, BIGT);

    if (!result)
      continue;

    if (debug)
    {
      hitPoints[posPtr++] = currentTask.outRay.origin;
      hitPoints[posPtr++] = result.point;
    }
    
    const Triangle triangle = triangles[result.triangleIdx];
    const Material material = materials[triangleMaterialIds[result.triangleIdx]];
    glm::fvec3 interpolatedNormal = triangle.normal(result.uv);

    unsigned int mask = INSIDE_BIT;

    if (glm::dot(interpolatedNormal, currentTask.outRay.direction) > 0.f)
      interpolatedNormal = -interpolatedNormal;  // We are inside an object. Flip the normal.
    else
      mask = 0x00000000; // We are outside. Unset bit.

    color += currentTask.filter * material.colorAmbient * 0.25f;

    const glm::fvec3 brightness = areaLightShading<SHADOWSAMPLING>(interpolatedNormal, light, bvh, result, triangles, curandState1, curandState2);
    color += currentTask.filter * material.colorDiffuse / glm::pi<float>() * brightness;

    if (material.shadingMode == material.GORAUD)
    {
      continue;
    }

    // Phong's specular highlight
    if ((mask & INSIDE_BIT) == 0x00 && material.shadingMode == material.PHONG)
    {
      const glm::fvec3 rm = reflectionDirection(interpolatedNormal, glm::normalize(light.getPosition() - result.point));
      color += material.colorSpecular * powf(__saturatef(glm::dot(rm, currentTask.outRay.direction)), material.shininess);
    }

    if (material.shadingMode == material.FRESNEL)
    {

      if (currentTask.levelsLeft == 0)
        continue;

      RaycastTask newTask; // Used twice for pushing

      mask = (material.colorSpecular.x != 0.f ||
          material.colorSpecular.y != 0.f ||
          material.colorSpecular.z != 0.f) ? REFLECTIVE_BIT | mask : mask;

      mask = (material.colorTransparent.x != 0.f ||
          material.colorTransparent.y != 0.f ||
          material.colorTransparent.z != 0.f) ? REFRACTIVE_BIT | mask : mask;

      float R = 1.f;

      if ((mask & REFRACTIVE_BIT) != 0x00) // Refractive
      {
        float idx1 = AIR_INDEX;
        float idx2 = material.refrIdx;

        float rat;

        if ((mask & INSIDE_BIT) != 0x00) // inside
          rat = __fdividef(idx1, idx2);
        else
          rat = __fdividef(idx2, idx1);

        // Something's not right here...

        // Transmittance and reflection according to fresnel
        const float cosi = fabsf(glm::dot(currentTask.outRay.direction, interpolatedNormal));

        if (sinf(acosf(cosi)) <= rat) // Check for total internal reflection
        {
          const float sin2t = fabs((idx1 / idx2) * (idx1 / idx2) * (1 - cosi * cosi));

          R = fresnelReflectioncoefficient(sin2t, cosi, idx1, idx2);

          const glm::fvec3 transOrig = result.point - interpolatedNormal * OFFSET_EPSILON;
          const glm::fvec3 transDir = refractionDirection(cosi, sin2t, interpolatedNormal, currentTask.outRay.direction, idx1, idx2);

          newTask.outRay = Ray(transOrig, transDir);
          newTask.levelsLeft = currentTask.levelsLeft - 1;
          newTask.filter = currentTask.filter * material.colorTransparent * (1 - R);
          stack[stackPtr] = newTask;
          ++stackPtr;
        }
      }

      if ((mask & REFLECTIVE_BIT) != 0x00) // Reflective
      {
        const glm::fvec3 reflOrig = result.point + interpolatedNormal * OFFSET_EPSILON;
        const glm::fvec3 reflDir = reflectionDirection(interpolatedNormal, currentTask.outRay.direction);

        newTask.outRay = Ray(reflOrig, reflDir);
        newTask.levelsLeft = currentTask.levelsLeft - 1;
        newTask.filter = currentTask.filter * material.colorSpecular * R;
        stack[stackPtr] = newTask;
        ++stackPtr;
      }
    }

  }

  return color;
}

template <const bool debug, typename curandStateType>
__device__ glm::fvec3 pathTrace(\
    const Node* bvh, \
    const Ray& ray, \
    const Triangle* triangles, \
    const Camera camera, \
    const Material* materials, \
    const unsigned int* triangleMaterialIds, \
    const Light light, \
    curandStateType& curandState1, \
    curandStateType& curandState2, \
    glm::fvec3* hitPoints = nullptr)
{
  unsigned int posPtr = 0;

  Ray currentRay = ray;
  glm::fvec3 color(0.f, 0.f, 0.f);
  glm::fvec3 throughput(1.f, 1.f, 1.f);

  float p = 1.0f;
  bool roulette = false;

  unsigned int bounces = PATH_TRACE_BOUNCES;
  bool terminate = false;
  unsigned int currentBounce = 0;

  do
  {
    const RaycastResult result = rayCast<debug, HitType::CLOSEST>(currentRay, bvh, triangles, BIGT);

    if (!result)
      return color;

    if (debug)
    {
      hitPoints[posPtr++] = currentRay.origin;
      hitPoints[posPtr++] = result.point;
    }

    const Triangle triangle = triangles[result.triangleIdx];
    const Material material = materials[triangleMaterialIds[result.triangleIdx]];
    glm::fvec3 interpolatedNormal = triangle.normal(result.uv);

    unsigned int mask = INSIDE_BIT;

    if (glm::dot(interpolatedNormal, currentRay.direction) > 0.f)
      interpolatedNormal = -interpolatedNormal;  // We are inside an object. Flip the normal.
    else
      mask = 0x00; // We are outside. Unset bit.

    color += throughput * material.colorAmbient * 0.25f;
    const glm::fvec3 brightness = areaLightShading<1>(interpolatedNormal, light, bvh, result, triangles, curandState1, curandState2);
    color += throughput * material.colorDiffuse / (glm::pi<float>() * p) * brightness;

    // Phong's specular highlight
    if ((mask & INSIDE_BIT) == 0x00 && material.shadingMode == material.PHONG)
    {
      const glm::fvec3 rm = reflectionDirection(interpolatedNormal, glm::normalize(light.getPosition() - result.point));
      color += material.colorSpecular * powf(__saturatef(glm::dot(rm, currentRay.direction)), material.shininess);
    }


    glm::fvec3 newDir, newOrig;
    glm::fmat3 B;

    if (material.shadingMode == material.FRESNEL)
    {
      mask = (material.colorSpecular.x != 0.f ||
          material.colorSpecular.y != 0.f ||
          material.colorSpecular.z != 0.f) ? REFLECTIVE_BIT | mask : mask;

      mask = (material.colorTransparent.x != 0.f ||
          material.colorTransparent.y != 0.f ||
          material.colorTransparent.z != 0.f) ? REFRACTIVE_BIT | mask : mask;

      float rP = 1.f; // Probability for reflection to occur. Depends on the strength of the specular and transparent colors.

      float R = 1.f; // Fresnel reflection coefficient
      float cosi, sin2t, idx1, idx2;

      if ((mask & REFRACTIVE_BIT) != 0x00)
      {
        float rLen = glm::length(material.colorSpecular);
        float tLen = glm::length(material.colorTransparent);

        rP = rLen / (rLen + tLen);

        idx1 = AIR_INDEX;
        idx2 = material.refrIdx;

        float rat;

        if ((mask & INSIDE_BIT) != 0x00) // inside
          rat = __fdividef(idx1, idx2);
        else
          rat = __fdividef(idx2, idx1);

        cosi = fabsf(glm::dot(currentRay.direction, interpolatedNormal));

        if (sinf(acosf(cosi)) <= rat) // Check for total internal reflection
        {
          sin2t = fabs((idx1 / idx2) * (idx1 / idx2) * (1 - cosi * cosi));
          R = fresnelReflectioncoefficient(sin2t, cosi, idx1, idx2);
        }
      }

      rP *= R;

      rP = rP / (rP + (1.f - rP) * (1.f - R));

      bool refl = hiprand_uniform(&curandState1) < rP;

      if (refl)
      {
        newDir = reflectionDirection(interpolatedNormal, currentRay.direction);
        newOrig = result.point + interpolatedNormal * OFFSET_EPSILON;
        throughput *= material.colorSpecular;
      }
      else
      {
        newDir = refractionDirection(cosi, sin2t, interpolatedNormal, currentRay.direction, idx1, idx2);
        newOrig = result.point - interpolatedNormal * OFFSET_EPSILON;
        throughput *= material.colorTransparent;
      }

    }
    else // Diffuse
    {
      B = getBasis(interpolatedNormal);

      do {
        newDir = glm::fvec3(hiprand_uniform(&curandState1) * 2.0f - 1.0f, hiprand_uniform(&curandState1) * 2.0f - 1.0f, 0.f);
      } while ((newDir.x * newDir.x + newDir.y * newDir.y) >= 1);

      newDir.z = glm::sqrt(1 - newDir.x * newDir.x - newDir.y * newDir.y);
      newDir = B * newDir;
      newDir = glm::normalize(newDir);

      newOrig = result.point + OFFSET_EPSILON * interpolatedNormal;

      p *= glm::dot(newDir, interpolatedNormal) * (1.f / glm::pi<float>());
      throughput *= material.colorDiffuse / glm::pi<float>() * glm::dot(newDir, interpolatedNormal);
    }

    currentRay = Ray(newOrig, newDir);

    if (currentBounce < bounces)
    {
      ++currentBounce;
    }
    else if (roulette)
    {
      ++currentBounce;
      p *= 0.8f; // Continuation probability
      terminate = hiprand_uniform(&curandState1) < 0.2f;
    }
    else
      terminate = true;

  } while (!terminate);


  return color;
}

template <typename hiprandState>
__global__ void initRand(const int seed, hiprandState* const curandStateDevPtr, const glm::ivec2 size)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= size.x || y >= size.y)
    return;

  hiprandState localState;
  hiprand_init(seed, x + y*size.x, 0, &localState);
  curandStateDevPtr[x + y * size.x] = localState;
}

__global__ void initRand(hiprandDirectionVectors64_t* sobolDirectionVectors, unsigned long long* sobolScrambleConstants, hiprandStateScrambledSobol64* state, const glm::ivec2 size)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  
  if (x >= size.x || y >= size.y)
    return;
    
  const unsigned int scrIdx = x + size.x * y;
  const unsigned int dirIdx = (x + size.x * y) % 10000;

  hiprandDirectionVectors64_t* dir = &sobolDirectionVectors[dirIdx];
  unsigned long long scr = sobolScrambleConstants[scrIdx];
  hiprandStateScrambledSobol64 localState;
    
  hiprand_init(*dir, scr, 0, &localState);

  state[x + size.x * y] = localState;
}

__device__ void writeToCanvas(const unsigned int x, const unsigned int y, const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize, const glm::vec3 data)
{
  const float4 out = make_float4(data.x, data.y, data.z, 1.f);
  surf2Dwrite(out, surfaceObj, (canvasSize.x - 1 - x) * sizeof(out), y);
  return;
}

__device__ glm::fvec3 readFromCanvas(const unsigned int x, const unsigned int y, const hipSurfaceObject_t& surfaceObj, const glm::ivec2 canvasSize)
{
  float4 in;
  surf2Dread(&in, surfaceObj, (canvasSize.x - 1 - x) * sizeof(in), y);

  const glm::fvec3 ret(in.x, in.y, in.z);

  return ret;
}

template <typename curandStateType>
__global__ void
cudaDebugRayTrace(\
    const glm::ivec2 pixelPos, \
    glm::fvec3* devPosPtr, \
    const glm::ivec2 size, \
    const Triangle* triangles, \
    const Camera camera, \
    const Material* materials, \
    const unsigned int* triangleMaterialIds, \
    const Light light, \
    curandStateType* curandStateDevXPtr, \
    curandStateType* curandStateDevYPtr, \
    const Node* bvh)
{
  const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(pixelPos.x, pixelPos.y, size);
  const float ar = (float) size.x / size.y;
  const Ray ray = camera.generateRay(nic, ar);

  (void) rayTrace<true>(\
      bvh,
      ray, \
      triangles, \
      camera, \
      materials, \
      triangleMaterialIds, \
      light, \
      curandStateDevXPtr[pixelPos.x + size.x * pixelPos.y], \
      curandStateDevYPtr[pixelPos.x + size.x * pixelPos.y], \
      devPosPtr);

  return;
}

template <typename curandStateType>
__global__ void
cudaDebugPathTrace(
    const glm::ivec2 pixelPos,
    glm::fvec3* devPosPtr,
    const glm::ivec2 size,
    const Triangle* triangles,
    const Camera camera,
    const Material* materials,
    const unsigned int* triangleMaterialIds,
    const Light light,
    curandStateType* curandStateDevXPtr,
    curandStateType* curandStateDevYPtr,
    const Node* bvh)
{
  const glm::fvec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(pixelPos.x, pixelPos.y, size);
  const float ar = (float) size.x / size.y;
  const Ray ray = camera.generateRay(nic, ar);

  (void) pathTrace<true>(
      bvh,
      ray,
      triangles,
      camera,
      materials,
      triangleMaterialIds,
      light,
      curandStateDevXPtr[pixelPos.x + size.x * pixelPos.y],
      curandStateDevYPtr[pixelPos.x + size.x * pixelPos.y],
      devPosPtr);

  return;
}

template <typename curandStateType>
__global__ void
__launch_bounds__(BLOCKWIDTH * BLOCKWIDTH, 24)
rayTraceKernel(\
    const hipSurfaceObject_t canvas, \
    const glm::ivec2 canvasSize, \
    const Triangle* triangles, \
    const Camera camera, \
    const Material* materials, \
    const unsigned int* triangleMaterialIds, \
    const Light light, \
    curandStateType* curandStateDevXPtr, \
    curandStateType* curandStateDevYPtr, \
    const Node* bvh)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  glm::vec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(x, y, canvasSize);

  Ray ray = camera.generateRay(nic, (float) canvasSize.x/canvasSize.y);

  curandStateType state1 = curandStateDevXPtr[x + y * canvasSize.x];
  curandStateType state2 = curandStateDevYPtr[x + y * canvasSize.x];

  glm::fvec3 color = rayTrace<false>(\
      bvh,
      ray, \
      triangles, \
      camera, \
      materials, \
      triangleMaterialIds, \
      light, \
      state1, \
      state2);

  curandStateDevXPtr[x + y * canvasSize.x] = state1;
  curandStateDevYPtr[x + y * canvasSize.x] = state2;

  writeToCanvas(x, y, canvas, canvasSize, color);

  return;
}

template <typename curandStateType>
__global__ void
pathTraceKernel(
    const unsigned int path,
    const hipSurfaceObject_t canvas,
    const glm::ivec2 canvasSize,
    const Triangle* triangles,
    const Camera camera,
    const Material* materials,
    const unsigned int* triangleMaterialIds,
    const Light light,
    curandStateType* curandStateDevXPtr,
    curandStateType* curandStateDevYPtr,
    const Node* bvh)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  glm::vec2 nic = camera.normalizedImageCoordinateFromPixelCoordinate(x, y, canvasSize);

  Ray ray = camera.generateRay(nic, (float) canvasSize.x/canvasSize.y);

  curandStateType state1 = curandStateDevXPtr[x + y * canvasSize.x];
  curandStateType state2 = curandStateDevYPtr[x + y * canvasSize.x];

  glm::fvec3 color = pathTrace<false>(\
      bvh,
      ray, \
      triangles, \
      camera, \
      materials, \
      triangleMaterialIds, \
      light, \
      state1, \
      state2);

  curandStateDevXPtr[x + y * canvasSize.x] = state1;
  curandStateDevYPtr[x + y * canvasSize.x] = state2;

  if (path == 1)
  {
    writeToCanvas(x, y, canvas, canvasSize, color);
  }
  else
  {
    const glm::fvec3 oldCol = readFromCanvas(x, y, canvas, canvasSize);
    const glm::fvec3 blend = oldCol * glm::fvec3((float) (path - 1) / path) + glm::fvec3((float) 1 / path) * color;
    writeToCanvas(x, y, canvas, canvasSize, blend);

  }

  return;
}

template <typename curandStateType>
__global__ void
cudaTestRnd(\
    const hipSurfaceObject_t canvas, \
    const glm::ivec2 canvasSize, \
    curandStateType* curandStateDevXPtr, \
    curandStateType* curandStateDevYPtr)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= canvasSize.x || y >= canvasSize.y)
    return;

  curandStateType localState1 = curandStateDevXPtr[x + y * canvasSize.x];
  curandStateType localState2 = curandStateDevYPtr[x + y * canvasSize.x];

  float r = hiprand_uniform(&localState1);
  float g = hiprand_uniform(&localState2);

  curandStateDevXPtr[x + y * canvasSize.x] = localState1;
  curandStateDevYPtr[x + y * canvasSize.x] = localState2;

  writeToCanvas(x, y, canvas, canvasSize, glm::fvec3(r, g, 0.f));

  return;
}

void CudaRenderer::reset()
{
  currentPath = 1;
}

void CudaRenderer::resize(const glm::ivec2& size)
{
  curandStateDevVecX.resize(size.x * size.y);
  curandStateDevVecY.resize(size.x * size.y);
  auto* curandStateDevXRaw = thrust::raw_pointer_cast(&curandStateDevVecX[0]);
  auto* curandStateDevYRaw = thrust::raw_pointer_cast(&curandStateDevVecY[0]);

  dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  dim3 grid( (size.x + block.x - 1) / block.x, (size.y + block.y - 1) / block.y);

  hipStream_t streams[2];
  CUDA_CHECK(hipStreamCreate(&streams[0]));
  CUDA_CHECK(hipStreamCreate(&streams[1]));

#ifdef QUASIRANDOM
  hiprandDirectionVectors64_t* hostDirectionVectors64;
  unsigned long long int* hostScrambleConstants64;
  
  hiprandDirectionVectors64_t* devDirectionVectors64;
  unsigned long long int* devScrambleConstants64;
  
  hiprandGetDirectionVectors64(&hostDirectionVectors64, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6);
  hiprandGetScrambleConstants64(&hostScrambleConstants64);
  
  CUDA_CHECK(hipMalloc((void **)&(devDirectionVectors64),             20000 * sizeof(hiprandDirectionVectors64_t)));
  CUDA_CHECK(hipMemcpy(devDirectionVectors64, hostDirectionVectors64, 20000 * sizeof(hiprandDirectionVectors64_t), hipMemcpyHostToDevice));
  
  CUDA_CHECK(hipMalloc((void **)&(devScrambleConstants64),              size.x * size.y * sizeof(unsigned long long int)));
  CUDA_CHECK(hipMemcpy(devScrambleConstants64, hostScrambleConstants64, size.x * size.y * sizeof(unsigned long long int), hipMemcpyHostToDevice));
  
  initRand<<<grid, block, 0, streams[0]>>>(devDirectionVectors64, devScrambleConstants64, curandStateDevXRaw, size);
  initRand<<<grid, block, 0, streams[1]>>>(devDirectionVectors64 + 10000, devScrambleConstants64, curandStateDevYRaw, size);
  
  CUDA_CHECK(hipFree(devDirectionVectors64));
  CUDA_CHECK(hipFree(devScrambleConstants64));

#else
  initRand<<<grid, block, 0, streams[0]>>>(0, curandStateDevXRaw, size);
  initRand<<<grid, block, 0, streams[1]>>>(5, curandStateDevYRaw, size);
#endif

  CUDA_CHECK(hipStreamDestroy(streams[0]));
  CUDA_CHECK(hipStreamDestroy(streams[1]));

  CUDA_CHECK(hipDeviceSynchronize());
}


CudaRenderer::CudaRenderer() : curandStateDevVecX(), curandStateDevVecY(), lastCamera(), lastSize(), currentPath(1)
{
  unsigned int cudaDeviceCount = 0;
  int cudaDevices[8];
  unsigned int cudaDevicesCount = 8;

	hipGLGetDevices(&cudaDeviceCount, cudaDevices, cudaDevicesCount, hipGLDeviceListCurrentFrame);

  if (cudaDeviceCount < 1)
  {
     std::cout << "No CUDA devices found" << std::endl;
     throw std::runtime_error("No CUDA devices available");
  }

  CUDA_CHECK(hipSetDevice(cudaDevices[0]));

  resize(glm::ivec2(WWIDTH, WHEIGHT));
}

CudaRenderer::~CudaRenderer()
{

}

void CudaRenderer::pathTraceToCanvas(GLTexture& canvas, const Camera& camera, GLModel& model, GLLight& light)
{
  if (model.getNTriangles() == 0)
    return;

  const glm::ivec2 canvasSize = canvas.getSize();
  const bool diffCamera = std::memcmp(&camera, &lastCamera, sizeof(Camera));
  const bool diffSize = (canvasSize != lastSize);

  if (diffCamera != 0 || diffSize != 0)
  {
    lastCamera = camera;
    lastSize = canvasSize;
    currentPath = 1;
  }

  auto* curandStateDevXRaw = thrust::raw_pointer_cast(&curandStateDevVecX[0]);
  auto* curandStateDevYRaw = thrust::raw_pointer_cast(&curandStateDevVecY[0]);

  auto surfaceObj = canvas.getCudaMappedSurfaceObject();
  const Triangle* devTriangles = model.getMappedCudaTrianglePtr();

  const dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  const dim3 grid( (canvasSize.x+ block.x - 1) / block.x, (canvasSize.y + block.y - 1) / block.y);

  pathTraceKernel<<<grid, block>>>(
      currentPath,
      surfaceObj,
      canvasSize,
      devTriangles,
      camera,
      model.getCudaMaterialsPtr(),
      model.getCudaTriangleMaterialIdsPtr(),
      light.getLight(),
      curandStateDevXRaw,
      curandStateDevYRaw,
      model.getDeviceBVH());

  ++currentPath;

  CUDA_CHECK(hipDeviceSynchronize());
  model.unmapCudaTrianglePtr();
  canvas.cudaUnmap();
}

void CudaRenderer::rayTraceToCanvas(GLTexture& canvas, const Camera& camera, GLModel& model, GLLight& light)
{
  if (model.getNTriangles() == 0)
    return;

  const glm::ivec2 canvasSize = canvas.getSize();

  auto* curandStateDevXRaw = thrust::raw_pointer_cast(&curandStateDevVecX[0]);
  auto* curandStateDevYRaw = thrust::raw_pointer_cast(&curandStateDevVecY[0]);

  auto surfaceObj = canvas.getCudaMappedSurfaceObject();
  const Triangle* devTriangles = model.getMappedCudaTrianglePtr();

  const dim3 block(BLOCKWIDTH, BLOCKWIDTH);
  const dim3 grid( (canvasSize.x+ block.x - 1) / block.x, (canvasSize.y + block.y - 1) / block.y);
  //dim3 block(BLOCKWIDTH * BLOCKWIDTH);
  //dim3 grid( (canvasSize.x * canvasSize.y + block.x - 1) / block.x);

  rayTraceKernel<<<grid, block>>>(\
      surfaceObj, \
      canvasSize, \
      devTriangles, \
      camera, \
      model.getCudaMaterialsPtr(), \
      model.getCudaTriangleMaterialIdsPtr(), \
      light.getLight(), \
      curandStateDevXRaw, \
      curandStateDevYRaw, \
      model.getDeviceBVH());

  //cudaTestRnd<<<grid, block>>>(surfaceObj, canvasSize, curandStateDevXRaw, curandStateDevYRaw);

  CUDA_CHECK(hipDeviceSynchronize());
  model.unmapCudaTrianglePtr();
  canvas.cudaUnmap();
}

std::vector<glm::fvec3> CudaRenderer::debugRayTrace(const glm::ivec2 pixelPos, const glm::ivec2 size, const Camera& camera, GLModel& model, GLLight& light)
{
  if (model.getNTriangles() == 0)
    return std::vector<glm::fvec3>();

  auto* curandStateDevXRaw = thrust::raw_pointer_cast(&curandStateDevVecX[0]);
  auto* curandStateDevYRaw = thrust::raw_pointer_cast(&curandStateDevVecY[0]);

  Triangle* devTriangles = model.getMappedCudaTrianglePtr();

  dim3 block(1, 1);
  dim3 grid(1, 1);

  unsigned int secondaryVertices = std::pow(2u, SECONDARY_RAYS) == 1 ? 0 : std::pow(2u, SECONDARY_RAYS) * 2;
  const int nVertices = (2 + secondaryVertices) * SHADOWSAMPLING;

  glm::fvec3* devPosPtr;
  CUDA_CHECK(hipMalloc((void**) &devPosPtr, nVertices * sizeof(glm::fvec3)));
  CUDA_CHECK(hipMemset((void*) devPosPtr, 0, nVertices * sizeof(glm::fvec3)));

  cudaDebugRayTrace<<<grid, block>>>(\
      size - pixelPos, \
      devPosPtr, \
      size, \
      devTriangles, \
      camera, \
      model.getCudaMaterialsPtr(), \
      model.getCudaTriangleMaterialIdsPtr(), \
      light.getLight(), \
      curandStateDevXRaw, \
      curandStateDevYRaw, \
      model.getDeviceBVH());

  CUDA_CHECK(hipDeviceSynchronize());
  model.unmapCudaTrianglePtr();

  std::vector<glm::fvec3> hitPos(nVertices);
  CUDA_CHECK(hipMemcpy(hitPos.data(), devPosPtr, nVertices * sizeof(glm::fvec3), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(devPosPtr));

  return hitPos;
}

std::vector<glm::fvec3> CudaRenderer::debugPathTrace(const glm::ivec2 pixelPos, const glm::ivec2 size, const Camera& camera, GLModel& model, GLLight& light)
{
  if (model.getNTriangles() == 0)
    return std::vector<glm::fvec3>();

  auto* curandStateDevXRaw = thrust::raw_pointer_cast(&curandStateDevVecX[0]);
  auto* curandStateDevYRaw = thrust::raw_pointer_cast(&curandStateDevVecY[0]);

  Triangle* devTriangles = model.getMappedCudaTrianglePtr();

  dim3 block(1, 1);
  dim3 grid(1, 1);

  const int nVertices = 2 * PATH_TRACE_BOUNCES;

  glm::fvec3* devPosPtr;
  CUDA_CHECK(hipMalloc((void**) &devPosPtr, nVertices * sizeof(glm::fvec3)));
  CUDA_CHECK(hipMemset((void*) devPosPtr, 0, nVertices * sizeof(glm::fvec3)));

  cudaDebugPathTrace<<<grid, block>>>(\
      size - pixelPos, \
      devPosPtr, \
      size, \
      devTriangles, \
      camera, \
      model.getCudaMaterialsPtr(), \
      model.getCudaTriangleMaterialIdsPtr(), \
      light.getLight(), \
      curandStateDevXRaw, \
      curandStateDevYRaw, \
      model.getDeviceBVH());

  CUDA_CHECK(hipDeviceSynchronize());
  model.unmapCudaTrianglePtr();

  std::vector<glm::fvec3> hitPos(nVertices);
  CUDA_CHECK(hipMemcpy(hitPos.data(), devPosPtr, nVertices * sizeof(glm::fvec3), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(devPosPtr));

  return hitPos;
}



